#include "hip/hip_runtime.h"
#include <assert.h>

#include "hip/hip_runtime.h"

void mallocOnCuda(DeviceMemoryDescr_t* device_memory_p, int padded_capacity){
    hipMalloc(&device_memory_p->prev_mesh_, padded_capacity * sizeof(double));
    hipMalloc(&device_memory_p->cur_mesh_, padded_capacity * sizeof(double));
    hipMalloc(&device_memory_p->next_mesh_, padded_capacity * sizeof(double));
}
void freeOnCuda(DeviceMemoryDescr_t* device_memory_p){
    hipFree(device_memory_p->prev_mesh_);
    hipFree(device_memory_p->cur_mesh_);
    hipFree(device_memory_p->next_mesh_);
}

/**
 * Get the index in the 1D array of size @rows * @columns * @z_columns,
 * that represents the 3D array [@rows; @columns; @z_columns]
 */
__device__ int packCudaIdx(int local_row_idx, int local_column_idx, int local_z_idx,
                           int rows, int columns, int z_columns){
    int packed_idx = local_row_idx * columns * z_columns +
                     local_column_idx * z_columns +
                     local_z_idx;
    assert(0 <= packed_idx && packed_idx < rows * columns * z_columns);
    return packed_idx;
}




/**
 * Compute the laplace operator over the current mesh cell.
 * @local_row_idx, @local_column_idx, @local_z_idx respects the padding.
 */
__device__ double laplace(const double* mesh,
                          int local_row_idx, int local_column_idx, int local_z_idx,
                          int padded_rows, int padded_columns, int padded_z_columns,
                          double spatial_step){
    double row_part = (mesh[packCudaIdx(local_row_idx - 1, local_column_idx, local_z_idx,
                                    padded_rows, padded_columns, padded_z_columns)] - 
                       2 * mesh[packCudaIdx(local_row_idx, local_column_idx, local_z_idx,
                                        padded_rows, padded_columns, padded_z_columns)] +
                       mesh[packCudaIdx(local_row_idx + 1, local_column_idx, local_z_idx,
                                        padded_rows, padded_columns, padded_z_columns)]) /
                      (spatial_step * spatial_step);
    double column_part = (mesh[packCudaIdx(local_row_idx, local_column_idx - 1, local_z_idx,
                                       padded_rows, padded_columns, padded_z_columns)] - 
                          2 * mesh[packCudaIdx(local_row_idx, local_column_idx, local_z_idx,
                                           padded_rows, padded_columns, padded_z_columns)] +
                          mesh[packCudaIdx(local_row_idx, local_column_idx + 1, local_z_idx,
                                       padded_rows, padded_columns, padded_z_columns)]) /
                         (spatial_step * spatial_step);
    double z_part = (mesh[packCudaIdx(local_row_idx, local_column_idx, local_z_idx - 1,
                                  padded_rows, padded_columns, padded_z_columns)] - 
                     2 * mesh[packCudaIdx(local_row_idx, local_column_idx, local_z_idx,
                                      padded_rows, padded_columns, padded_z_columns)] +
                     mesh[packCudaIdx(local_row_idx, local_column_idx, local_z_idx + 1,
                                  padded_rows, padded_columns, padded_z_columns)]) /
                    (spatial_step * spatial_step);
    return row_part + column_part + z_part;
}
/**
 * A kernel for the numerical method.
 * Warning:
 *      Doesn't check if the index is boundary for the global mesh,
 *      fills these indexes with incorrect values.
 *      Compute global boundary elements only after calling this kernel.
 */
__global__ void methodIterKernel(const double* prev_mesh, const double* cur_mesh, double* next_mesh,
                                 int rows, int columns, int z_columns,
                                 double tau, double h){

    // A grid-striding loop over the flattened 3D mesh
    for(int row_idx = 1; row_idx <= rows; ++row_idx){
        for(int column_idx = 1 + blockIdx.x; column_idx <= columns; column_idx += gridDim.x){
            for(int z_idx = 1 + threadIdx.x; z_idx <= z_columns; z_idx += blockDim.x){
                int packed_idx = packCudaIdx(row_idx, column_idx, z_idx,
											 rows + 2, columns + 2, z_columns + 2);
                next_mesh[packed_idx] = tau * tau * laplace(cur_mesh, row_idx, column_idx, z_idx,
                                                            rows + 2, columns + 2, z_columns + 2, h)
                                        - prev_mesh[packed_idx] + 2 * cur_mesh[packed_idx];
            }
        }
    }
}

/**
 * A wrapper for the kernel call.
 * Copies the memory to device
 */
void callMethodIterKernel(DeviceMemoryDescr_t* device_memory_p,
                          HostMemoryDescr_t* host_memory_p,
                          int padded_capacity,
                          int rows, int columns, int z_columns,
                          double tau, double h){
    hipMemcpy(device_memory_p->prev_mesh_, host_memory_p->prev_mesh_, padded_capacity * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(device_memory_p->cur_mesh_, host_memory_p->cur_mesh_, padded_capacity * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(device_memory_p->next_mesh_, host_memory_p->next_mesh_, padded_capacity * sizeof(double),
               hipMemcpyHostToDevice);
    int threads_per_block = 1024;
    int blocks_per_grid = 2;
    methodIterKernel<<<blocks_per_grid, threads_per_block>>>(device_memory_p->prev_mesh_, device_memory_p->cur_mesh_, device_memory_p->next_mesh_,
                                                             rows, columns, z_columns, tau, h);
    hipMemcpy(host_memory_p->next_mesh_, device_memory_p->next_mesh_, padded_capacity * sizeof(double),
               hipMemcpyDeviceToHost);
}